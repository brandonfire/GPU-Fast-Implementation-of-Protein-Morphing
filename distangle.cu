#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
//#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "input.h"
//#include "intermed.h"
#define DISTANCERSIZE 15625
typedef struct dist_recorder{ //structure to record distance of each position we can use index to indicate poisiton
	//int pos1;//position 1
	//int pos2;//position 2
	double d;//this is the distance
} Distancer;

typedef struct ang_recorder{ //structure to record angler of each position
	//int pos;//position use index as pos
	double d;//this is the angle
} Angler;

Proteinbone * protein;
Distancer * distrecorder;
Angler * anglecorder;

/*device function to caculate distance*/
__device__
double d_p2p_distance(double x1, double x2, double y1, double y2, double z1, double z2) {
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}

/* chengbin: kenrel function to caculate histogram*/


__global__
void D_dist_ang_function(double * x,double *y, double * z, Distancer * distrecorder, Angler * anglerecorder, int proteinlen){
	/*shared tiling input*/
	__shared__ double ix1[34];
	__shared__ double iy1[34];
	__shared__ double iz1[34];
	__shared__ double ix2[34];
	__shared__ double iy2[34];
	__shared__ double iz2[34];

	/*shared ouput
	extern __shared__ unsigned long long p_hist[];*/
	int i, j, ti,k;
	//input[threadIdx.x]=atomlist[];
	int gd =gridDim.x;
	int bd = blockDim.x;
	int bdx = blockIdx.x;
	ti = threadIdx.x;
	i = bdx * bd + ti;
	int didx;
	
	//for(j=ti;j<n_buckets;j+=bd)p_hist[j]=0;//iniatilize the ouput histogram
	//copy the anchor tile data to ix1,iy1,iz1 according to i
	if(i<proteinlen){	
	ix1[ti] = x[i];
	iy1[ti] = y[i];
	iz1[ti] = z[i];	
	}
	if(i=proteinlen-1){
		if(bdx!=0){
			ix1[32] = x[(bdx+1) * bd];
			iy1[32] = y[(bdx+1) * bd];
			iz1[32] = z[(bdx+1) * bd];
			ix1[33] = x[(bdx-1) * bd+31];
			iy1[33] = y[(bdx-1) * bd+31];
			iz1[33] = z[(bdx-1) * bd+31];		
		}else{
			ix1[32] = x[(bdx+1) * bd];
			iy1[32] = y[(bdx+1) * bd];
			iz1[32] = z[(bdx+1) * bd];
		}	
	}
	//double COS_70;
	//double COS_120;
	//COS_70    = cos(70. * 3.15159 / 180.);
    	//COS_120   = -0.5; 
	//angle part
	int i_1;
	if(ti == 0){
	i_1 = 33;} else {i_1=ti-1;}
	double aa = d_p2p_distance(ix1[ti],ix1[i_1],iy1[ti],iy1[i_1],iz1[ti],iz1[i_1]);
   	double bb = d_p2p_distance(ix1[ti],ix1[ti+1],iy1[ti],iy1[ti+1],iz1[ti],iz1[ti+1]);
   	double cc = d_p2p_distance(ix1[i_1],ix1[ti+1],iy1[i_1],iy1[ti+1],iz1[i_1],iz1[ti+1]);

   	double dd = 2*sqrt(aa*bb);

   	double cos_v1v2v3 = (aa + bb - cc)/dd;
	anglerecorder[i].d = cos_v1v2v3;
	

	//ix2[ti] = x[i];
	//iy2[ti] = y[i];
	//iz2[ti] = z[i];
	__syncthreads();
	//distance part
	double dist;
	
	int lastblock = gd-1;
	int lastblocklength = proteinlen - bd*(gd-1);
	if(bdx<lastblock)
	{
		
		for(j=ti+1; j<bd;j++)
		{
			dist = d_p2p_distance(ix1[ti],ix1[j],iy1[ti],iy1[j],iz1[ti],iz1[j]);
			//d_pos = (int) (dist / PDH_w);
			//atomicAdd(&(p_hist[d_pos]),1);
			didx = i*125+(bdx * bd + j);
			distrecorder[didx].d = dist;
		}
		

		
	__syncthreads();
	} else 
	{
		
		if(i<proteinlen)
		{

			
			
			for(j=ti+1; j<lastblocklength;j++)
			{
				dist = d_p2p_distance(ix1[ti],ix1[j],iy1[ti],iy1[j],iz1[ti],iz1[j]);
				//d_pos = (int) (dist / PDH_w);
				//atomicAdd(&(p_hist[d_pos]),1);
				didx = i*125+(bdx * bd + j);
				distrecorder[didx].d = dist;
			}
		}
		__syncthreads();
	}
	__syncthreads();
	
	//calcute the points between blocks.
	int cycle = ceil(gd/2.0);//becareful the last block	
	for(k=1;k<cycle;k++)//caculate points between blocks
		{
			j = (bdx+k)%gd;
			if(j<lastblock) // j is not the last block
			{
				ix2[ti] = x[j* bd + ti];
				iy2[ti] = y[j* bd + ti];
				iz2[ti] = z[j* bd + ti];
				__syncthreads();
				if(i<proteinlen)
				{
					for(int m = 0; m<bd; m++)
					{
						dist = d_p2p_distance(ix1[ti],ix2[m],iy1[ti],iy2[m],iz1[ti],iz2[m]);
						//d_pos = (int) (dist / PDH_w);
						//atomicAdd(&(p_hist[d_pos]),1);
						didx = i*125+ j* bd + m;
						distrecorder[didx].d = dist;
					}
				}
				__syncthreads();
			} else //J is the last block
			{
				
				if(ti<lastblocklength)
				{
					ix2[ti] = x[j* bd + ti];
					iy2[ti] = y[j* bd + ti];
					iz2[ti] = z[j* bd + ti];
				}
				__syncthreads();
				if(i<proteinlen)
				{
					for(int m = 0; m<lastblocklength; m++)
					{
						dist = d_p2p_distance(ix1[ti],ix2[m],iy1[ti],iy2[m],iz1[ti],iz2[m]);
						//d_pos = (int) (dist / PDH_w);
						//atomicAdd(&(p_hist[d_pos]),1);
						didx = i*125+ j* bd + m;
						distrecorder[didx].d = dist;
					}
				}
				__syncthreads();
			}
	
		}//last half cycle for gridDim.x%2==0
		if(gd%2==0)
		{
			
			if(bdx<gd/2)
			{
				j = (bdx+cycle)%gd;
				if(j<lastblock) // j is not the last block
				{
					ix2[ti] = x[j* bd + ti];
					iy2[ti] = y[j* bd + ti];
					iz2[ti] = z[j* bd + ti];
					__syncthreads();
					if(i<proteinlen)
					{
						for(int m = 0; m<bd; m++)
						{
							dist = d_p2p_distance(ix1[ti],ix2[m],iy1[ti],iy2[m],iz1[ti],iz2[m]);
							//d_pos = (int) (dist / PDH_w);
							//atomicAdd(&(p_hist[d_pos]),1);
							didx = i*125+ j* bd + m;
							distrecorder[didx].d = dist;
						}
					}
					__syncthreads();
				} else //J is the last block
				{
					
					if(ti<lastblocklength)
					{
						ix2[ti] = x[j* bd + ti];
						iy2[ti] = y[j* bd + ti];
						iz2[ti] = z[j* bd + ti];
						
					}
					__syncthreads();
					if(i<proteinlen)
					{
						for(int m = 0; m<lastblocklength; m++)
						{
							dist = d_p2p_distance(ix1[ti],ix2[m],iy1[ti],iy2[m],iz1[ti],iz2[m]);
							//d_pos = (int) (dist / PDH_w);
							//atomicAdd(&(p_hist[d_pos]),1);
							didx = i*125+ j* bd + m;
							distrecorder[didx].d = dist;
						}
					
					}
					__syncthreads();
				}



			}

		}

	__syncthreads();
	
	


}

void pdbtoarray(char *file_name,Proteinbone *atom)//function that will read pdb file and store to array
{
	//for pure proccessed pdb I only read XYZ to my array
	char Xtmp[9] = { ' ', ' ',' ', ' ', ' ', ' ', ' ', ' ' };
	char Ytmp[9] = { ' ', ' ',' ', ' ', ' ', ' ', ' ', ' ' };
	char Ztmp[9] = { ' ', ' ',' ', ' ', ' ', ' ', ' ', ' ' };
	char fileBuffer[RAMusage]; // input character array from file
	double X, Y, Z;// output floats
	int line, i, j, endOfLine;//, endOfComment, size; // logical integers
	FILE *fileID;
	fileID = fopen( file_name, "r" );
	if ( fileID != NULL ) 
	{	
		line = 0;
		while ( fgets( fileBuffer, sizeof (fileBuffer), fileID ) != NULL )
		{
			for ( i = 0; fileBuffer[i] != '\0' ; i++ ) {} endOfLine = i - 1;// find end of line
			if (endOfLine > 54  )
			{
				j = 0; for ( i = 30; i < 38; i++ ) { Xtmp[j]      = fileBuffer[i]; j++; }
				j = 0; for ( i = 38; i < 46; i++ ) { Ytmp[j]      = fileBuffer[i]; j++; }
				j = 0; for ( i = 46; i < 54; i++ ) { Ztmp[j]      = fileBuffer[i]; j++; }
				X     = atof( Xtmp );
				Y     = atof( Ytmp );
				Z     = atof( Ztmp );
				atom[line].x_pos = X;
				atom[line].y_pos = Y;
				atom[line].z_pos = Z;
				//printf("%8.3lf%8.3lf%8.3lf\n", X, Y, Z);
				line++;
			}
		}
	}else { printf("File could not be opened\n"); } // if file opening failed
	fclose( fileID );
}

/*
__global__
void D_initialize(bucket * h, int n_buckets){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<n_buckets) h[i].d_cnt=0;

}*/
int main(int argc, char **argv)
{
	if(argc != 2) {
        printf("ERROR please input one argument: %s {the input pdb file}\n",argv[0]);
        exit(1);
    	}
    	protein = (Proteinbone *)malloc(sizeof(Proteinbone)*PROTEINLENGTH);
    	pdbtoarray(argv[1],protein);
    	double * h_x, * h_y, * h_z;//seperate host input array
    	double * d_x,* d_y,* d_z;//seperate device input array	
    	h_x = (double *)malloc(sizeof(double)*PROTEINLENGTH);
	h_y = (double *)malloc(sizeof(double)*PROTEINLENGTH);
	h_z = (double *)malloc(sizeof(double)*PROTEINLENGTH);
	hipMalloc((void**)&d_x, sizeof(double)*PROTEINLENGTH);
	hipMalloc((void**)&d_y, sizeof(double)*PROTEINLENGTH);	
	hipMalloc((void**)&d_z, sizeof(double)*PROTEINLENGTH);
	int i;
	for(i = 0;  i < PROTEINLENGTH; i++) {
		h_x[i] = protein[i].x_pos;
		h_y[i] = protein[i].y_pos;
		h_z[i] = protein[i].z_pos;
	}
	distrecorder = (Distancer *)malloc(sizeof(Distancer)*DISTANCERSIZE);
	anglecorder = (Angler *)malloc(sizeof(Angler)*PROTEINLENGTH);
	Distancer * d_distrecorder;
	Angler * d_anglerecorder;
	hipMalloc((void**)&d_distrecorder, sizeof(Distancer)*DISTANCERSIZE);
	hipMalloc((void**)&d_anglerecorder, sizeof(Angler)*PROTEINLENGTH);	
	hipMemcpy(d_x,h_x,sizeof(double)*PROTEINLENGTH, hipMemcpyHostToDevice);
	hipMemcpy(d_y,h_y,sizeof(double)*PROTEINLENGTH, hipMemcpyHostToDevice);
	hipMemcpy(d_z,h_z,sizeof(double)*PROTEINLENGTH, hipMemcpyHostToDevice);
	/*chengbin: defince grid and block parameter*/
	dim3 dimGrid((int)ceil(PROTEINLENGTH/(float)32),1,1);
	dim3 dimBlock(32,1,1);
	D_dist_ang_function<<<dimGrid,dimBlock>>>(d_x, d_y, d_z, d_distrecorder,d_anglerecorder, PROTEINLENGTH);
	hipMemcpy(distrecorder,d_distrecorder,sizeof(Distancer)*DISTANCERSIZE, hipMemcpyDeviceToHost);
	hipMemcpy(anglecorder,d_anglerecorder,sizeof(Angler)*PROTEINLENGTH, hipMemcpyDeviceToHost);
	for(i = 0; i < DISTANCERSIZE; i ++ ){

	printf("%f\n",distrecorder[i].d);
	}
	for(i = 0; i < PROTEINLENGTH; i ++ ){

	printf("angle at %d: %f\n",i,anglecorder[i].d);
	}


	hipFree(d_distrecorder);
	hipFree(d_anglerecorder);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	free(protein);
	free(anglecorder);
	free(h_x);
	free(h_y);
	free(h_z);	
	free(distrecorder);
	return 0;

}

